#include "hip/hip_runtime.h"
#include <iostream>
#include "CusparseCSR.cuh"

namespace CusparseCSRTests
{
	CusparseCSR buildMatrix()
	{
		int order = 6;
		int nnz = 19;
		double* values = new double[nnz] { 10, -2, 3, 9, 3, 7, 8, 7, 3, 8, 7, 5, 8, 9, 9, 13, 4, 2, -1 };
		int* colIndexes = new int[nnz] { 0, 4, 0, 1, 5, 1, 2, 3, 0, 2, 3, 4, 1, 3, 4, 5, 1, 4, 5 };
		int* rowPointers = new int[order + 1]{ 0, 2, 5, 8, 12, 16, 19 };
		return CusparseCSR(order, nnz, values, colIndexes, rowPointers);
	}

	void printMatrix()
	{
		CusparseCSR matrix = buildMatrix();
		std::cout << matrix;
	}

	void checkResult(double* result)
	{
		int order = 6;
		double expected[]{ 12, -57, 98, 145, -63, 8 };
		double tolerance = 1e-6;
		bool isCorrect = true;
		for (int i = 0; i < order; ++i)
		{
			if (abs(result[i] / expected[i] - 1.0) > tolerance)
			{
				isCorrect = false;
				break;
			}
		}
		if (isCorrect) std::cout << "SpMV was correct.\n";
		else std::cout << "Error in SpMV!\n";
	}

	void CSRspMVTest()
	{
		CusparseCSR matrix = buildMatrix();
		double* x = new double[6]{ 2, -3, 7, 9, 4, -12 };
		double* y = new double[6];
		matrix.spMV(x, y);
		checkResult(y);
	}
};
