#include <iostream>
#include <exception>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include "CusparseCSR.cuh"
#include "CSR.hpp"

CusparseCSR::CusparseCSR(int order, int nnz, double* values, int* columnIndexes, int* rowPointers):
	h_order(order), h_nnz(nnz), d_values(nullptr), d_columnIndices(nullptr), d_rowPointers(nullptr)
{
	hipError_t err;
	err = hipMalloc((void**)&d_values, nnz * sizeof(double));
	err = hipMemcpy(d_values, values, nnz * sizeof(double), hipMemcpyHostToDevice);
	err = hipMalloc((void**)&d_columnIndices, nnz * sizeof(int));
	err = hipMemcpy(d_columnIndices, columnIndexes, nnz * sizeof(int), hipMemcpyHostToDevice);
	err = hipMalloc((void**)&d_rowPointers, (order+1) * sizeof(int));
	err = hipMemcpy(d_rowPointers, rowPointers, (order + 1) * sizeof(int), hipMemcpyHostToDevice);

	delete[] values;
	values = nullptr;
	delete[] columnIndexes;
	columnIndexes = nullptr;
	delete[] rowPointers;
	rowPointers = nullptr;
}

CusparseCSR::~CusparseCSR()
{
	hipError_t err;
	err = hipFree(d_values);
	err = hipFree(d_columnIndices);
	err = hipFree(d_rowPointers);
}

int CusparseCSR::getOrder() const
{
	return h_order;
}

int CusparseCSR::getNonZeroCount() const
{
	return h_nnz;
}

void CusparseCSR::spMV(double* lhs, double* rhs) const
{
}

void CusparseCSR::spMV_T(double* lhs, double* rhs) const
{
	throw std::exception("Not implemented yet");
}

CSR CusparseCSR::toCSR() const
{
	hipError_t err;
	double* hValues = new double[h_nnz];
	err = hipMemcpy(hValues, d_values, h_nnz * sizeof(double), hipMemcpyDeviceToHost);
	int* hColIndices = new int[h_nnz];
	err = hipMemcpy(hColIndices, d_columnIndices, h_nnz * sizeof(int), hipMemcpyDeviceToHost);
	int* hRowPtrs = new int[h_order + 1];
	err = hipMemcpy(hRowPtrs, d_rowPointers, (h_order + 1) * sizeof(int), hipMemcpyDeviceToHost);
	return CSR(h_order, h_nnz, hValues, hColIndices, hRowPtrs);
}

std::ostream& operator<<(std::ostream& out, const CusparseCSR& matrix)
{
	return out << matrix.toCSR();
}

std::ostream& operator<<(std::ostream& out, const CusparseCSR* matrix)
{
	return out << matrix->toCSR();
}