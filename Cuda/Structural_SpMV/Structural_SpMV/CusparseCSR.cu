#include <iostream>
#include <exception>
#include <hip/hip_runtime.h>

#include "CusparseCSR.cuh"
#include "CSR.hpp"

CusparseCSR::CusparseCSR(int order, int nnz, double* values, int* columnIndexes, int* rowPointers):
	_order(order), _nnz(nnz)
{
	hipError_t err;
	err = hipMalloc((void**)&d_values, nnz * sizeof(double));
	err = hipMemcpy(d_values, values, nnz * sizeof(double), hipMemcpyHostToDevice);
	err = hipMalloc((void**)&d_columnIndices, nnz * sizeof(int));
	err = hipMemcpy(d_columnIndices, columnIndexes, nnz * sizeof(int), hipMemcpyHostToDevice);
	err = hipMalloc((void**)&d_rowPointers, (order+1) * sizeof(int));
	err = hipMemcpy(d_rowPointers, rowPointers, (order + 1) * sizeof(int), hipMemcpyHostToDevice);

	delete[] values;
	values = nullptr;
	delete[] columnIndexes;
	columnIndexes = nullptr;
	delete[] rowPointers;
	rowPointers = nullptr;

	hipsparseCreateMatDescr(&_description); // General matrix, index base = 0
	d_ZERO = nullptr;
	d_ONE = nullptr;
}

CusparseCSR::~CusparseCSR()
{
	hipsparseDestroyMatDescr(_description);
	hipError_t err;
	err = hipFree(d_values);
	err = hipFree(d_columnIndices);
	err = hipFree(d_rowPointers);
}

int CusparseCSR::getOrder() const
{
	return _order;
}

int CusparseCSR::getNonZeroCount() const
{
	return _nnz;
}

// y = alpha * A * x + beta * y
void CusparseCSR::spMV(hipsparseHandle_t handle, double* dLhs, double* dRhs) const
{
	double alpha = 1.0;
	double beta = 0.0;
	hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, _order, _order, _nnz, &alpha, _description, 
		d_values, d_rowPointers, d_columnIndices, dLhs, &beta, dRhs);
	hipDeviceSynchronize();
}

void CusparseCSR::spMV_T(hipsparseHandle_t handle, double* dLhs, double* dRhs) const
{
	double alpha = 1.0;
	double beta = 0.0;
	hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, _order, _order, _nnz, &alpha, _description,
		d_values, d_rowPointers, d_columnIndices, dLhs, &beta, dRhs);
	hipDeviceSynchronize();
}

CSR CusparseCSR::toCSR() const
{
	hipError_t err;
	double* hValues = new double[_nnz];
	err = hipMemcpy(hValues, d_values, _nnz * sizeof(double), hipMemcpyDeviceToHost);
	int* hColIndices = new int[_nnz];
	err = hipMemcpy(hColIndices, d_columnIndices, _nnz * sizeof(int), hipMemcpyDeviceToHost);
	int* hRowPtrs = new int[_order + 1];
	err = hipMemcpy(hRowPtrs, d_rowPointers, (_order + 1) * sizeof(int), hipMemcpyDeviceToHost);
	return CSR(_order, _nnz, hValues, hColIndices, hRowPtrs);
}

std::ostream& operator<<(std::ostream& out, const CusparseCSR& matrix)
{
	return out << matrix.toCSR();
}

std::ostream& operator<<(std::ostream& out, const CusparseCSR* matrix)
{
	return out << matrix->toCSR();
}