
#include <hip/hip_runtime.h>
#include <iostream>

int testKernel();
namespace DOKTests { void buildAndPrint(); void testSlicing(); void testConversionToCSR(); }
namespace CSRTests { void spMVTest(); }
namespace CusparseCSRTests { void cusparseTest(); void printMatrix(); }
namespace AbaqusReaderTests { void ReadMatrix(); }
namespace BenchmarkTests { void runBenchmark(); }

int main()
{
	//testKernel();
	//DOKTests::testSlicing();
	//DOKTests::testConversionToCSR();
	//CSRTests::spMVTest();
	//CusparseCSRTests:cusparseTest();
	CusparseCSRTests::printMatrix();
	//AbaqusReaderTests::ReadMatrix();
	//BenchmarkTests::runBenchmark();
	

	std::cout << "\n\nPress any key to exit: ";
	char a;
	std::cin >> a;
	return 0;
}