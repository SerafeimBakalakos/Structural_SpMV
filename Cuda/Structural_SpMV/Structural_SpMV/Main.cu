
#include <hip/hip_runtime.h>
#include <iostream>

int testKernel();
namespace DOKTests { void buildAndPrint(); void testSlicing(); void testConversionToCSR(); }
namespace CSRTests { void spMVTest(); }
namespace CusparseCSRTests {
	void cusparseTest(); void printMatrix(); void spMVTest(); }
namespace AbaqusReaderTests { void ReadMatrix(); }
namespace BenchmarkTests { void runBenchmark(); }

int main()
{
	//testKernel();
	//DOKTests::testSlicing();
	//DOKTests::testConversionToCSR();
	//CSRTests::spMVTest();
	//CusparseCSRTests:cusparseTest();
	CusparseCSRTests::spMVTest();
	//AbaqusReaderTests::ReadMatrix();
	//BenchmarkTests::runBenchmark();
	

	std::cout << "\n\nPress any key to exit: ";
	char a;
	std::cin >> a;
	return 0;
}