
#include <hip/hip_runtime.h>
#include <iostream>

int testKernel();
void cusparseTest();
void buildAndPrint();
void testSlicing();
void testConversionToCSR();
void ReadMatrix();
void CSRspMVTest();
void runBenchmark();

namespace CusparseCSRTests
{
	void printMatrix();
}

int main()
{
	//testKernel();
	//cusparseTest();
	//testSlicing();
	//ReadMatrix();
	//CSRspMVTest();
	//testConversionToCSR();
	//runBenchmark();
	CusparseCSRTests::printMatrix();

	std::cout << "\n\nPress any key to exit: ";
	char a;
	std::cin >> a;
	return 0;
}