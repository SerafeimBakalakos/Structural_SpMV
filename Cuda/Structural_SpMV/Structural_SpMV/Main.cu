
#include <hip/hip_runtime.h>
#include <iostream>

int testKernel();
namespace DOKTests { void buildAndPrint(); void testSlicing(); void testConversionToCSR(); void testConversionToELL(); }
namespace CSRTests { void spMVTest(); }
namespace ELLTests { void buildAndPrintMatrix(); void spMVTest(); }
namespace CusparseCSRTests { void cusparseTest(); void printMatrix(); void spMVTest(); }
namespace AbaqusReaderTests { void ReadMatrix(); }
namespace BenchmarkTests { void runBenchmark(); }

int main()
{
	//testKernel();
	//DOKTests::testSlicing();
	//DOKTests::testConversionToCSR();
	DOKTests::testConversionToELL();
	//CSRTests::spMVTest();
	//ELLTests::buildAndPrintMatrix();
	//ELLTests::spMVTest();
	//CusparseCSRTests:cusparseTest();
	//CusparseCSRTests::spMVTest();
	//AbaqusReaderTests::ReadMatrix();
	//BenchmarkTests::runBenchmark();
	

	std::cout << "\n\nPress any key to exit: ";
	char a;
	std::cin >> a;
	return 0;
}